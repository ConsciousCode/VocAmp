
#include <hip/hip_runtime.h>
__global__ void vis2hid(float* vis,float* w8s,float* hid){
	hid[threadIdx.x]+=vis[threadIdx.y]*w8s[threadIdx.y*blockDim.x+threadIdx.x];
}

__global__ void hid2vis(float* hid,float* w8s,float* vis){
	vis[threadIdx.y]+=hid[threadIdx.x]*w8s[threadIdx.y*blockDim.x+threadIdx.x];
}

__global__ void learning(
		float* w8s,float* posv,float* posh,float* negv,float* negh
){
	w8s[threadIdx.y*blockDim.x+threadIdx.x]+=
		posv[threadIdx.y]*posh[threadIdx.x]-negv[threadIdx.y]*negh[threadIdx.x];
}

void deconstruct(float* vis,float* w8s,float* hid,unsigned nv,unsigned nh){
	memset(hid,0,nh*sizeof(float));
	vis2hid<<<nv,nh>>>(vis,w8s,hid);
}

void reconstruct(float* vis,float* w8s,float* hid,unsigned nv,unsigned nh){
	memset(vis,0,nv*sizeof(float));
	hid2vis<<<nv,nh>>>(vis,w8s,hid);
}
